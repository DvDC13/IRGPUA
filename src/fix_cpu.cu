#include "hip/hip_runtime.h"
#include "fix_cpu.cuh"
#include "image.hh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>

void fix_image_cpu(Image& to_fix)
{
    const int image_size = to_fix.width * to_fix.height;

    // #1 Compact

    // Build predicate vector

    std::vector<int> predicate(to_fix.buffer.size(), 0);

    constexpr int garbage_val = -27;
    for (int i = 0; i < to_fix.buffer.size(); ++i)
        if (to_fix.buffer[i] != garbage_val)
            predicate[i] = 1;

    // Compute the exclusive sum of the predicate

    std::exclusive_scan(predicate.begin(), predicate.end(), predicate.begin(), 0);

    // Scatter to the corresponding addresses

    for (std::size_t i = 0; i < predicate.size(); ++i)
        if (to_fix.buffer[i] != garbage_val)
            to_fix.buffer[predicate[i]] = to_fix.buffer[i];
    
    // #2 Apply map to fix pixels

    for (int i = 0; i < image_size; ++i)
    {
        if (i % 4 == 0)
            to_fix.buffer[i] += 1;
        else if (i % 4 == 1)
            to_fix.buffer[i] -= 5;
        else if (i % 4 == 2)
            to_fix.buffer[i] += 3;
        else if (i % 4 == 3)
            to_fix.buffer[i] -= 8;
    }

    // #3 Histogram equalization

    // Histogram

    std::array<int, 256> histo;
    histo.fill(0);
    for (int i = 0; i < image_size; ++i)
        ++histo[to_fix.buffer[i]];

    // Compute the inclusive sum scan of the histogram

    std::inclusive_scan(histo.begin(), histo.end(), histo.begin());;

    // Find the first non-zero value in the cumulative histogram

    auto first_none_zero = std::find_if(histo.begin(), histo.end(), [](auto v) { return v != 0; });

    const int cdf_min = *first_none_zero;

    // Apply the map transformation of the histogram equalization

    std::transform(to_fix.buffer.data(), to_fix.buffer.data() + image_size, to_fix.buffer.data(),
        [image_size, cdf_min, &histo](int pixel)
            {
                return std::roundf(((histo[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
            }
    );
}

int main_cpu([[maybe_unused]] int argc, [[maybe_unused]] char** argv, Pipeline& pipeline)
{
    // -- Main loop containing image retring from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::vector<Image> images(nb_images);

    // - One CPU thread is launched for each image

    std::cout << "Done, starting compute" << std::endl;

    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        // TODO : make it GPU compatible (aka faster)
        // You will need to copy images one by one on the GPU
        // You can store the images the way you want on the GPU
        // But you should treat the pipeline as a pipeline :
        // You *must not* copy all the images and only then do the computations
        // You must get the image from the pipeline as they arrive and launch computations right away
        // There are still ways to speeds this process of course (wait for last class)
        images[i] = pipeline.get_image(i);
        fix_image_cpu(images[i]);
    }

    std::cout << "Done with compute, starting stats" << std::endl;

    // -- All images are now fixed : compute stats (total then sort)

    // - First compute the total of each image

    // TODO : make it GPU compatible (aka faster)
    // You can use multiple CPU threads for your GPU version using openmp or not
    // Up to you :)
    #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        auto& image = images[i];
        const int image_size = image.width * image.height;

        image.to_sort.total = std::reduce(image.buffer.begin(), image.buffer.begin() + image_size, 0);
    }

    // - All totals are known, sort images accordingly (OPTIONAL)
    // Moving the actual images is too expensive, sort image indices instead
    // Copying to an id array and sort it instead

    // TODO OPTIONAL : for you GPU version you can store it the way you want
    // But just like the CPU version, moving the actual images while sorting will be too slow
    using ToSort = Image::ToSort;
    std::vector<ToSort> to_sort(nb_images);
    std::generate(to_sort.begin(), to_sort.end(), [n = 0, images] () mutable
    {
        return images[n++].to_sort;
    });

    // TODO OPTIONAL : make it GPU compatible (aka faster)
    std::sort(to_sort.begin(), to_sort.end(), [](ToSort a, ToSort b) {
        return a.total < b.total;
    });

    // TODO : Test here that you have the same results
    // You can compare visually and should compare image vectors values and "total" values
    // If you did the sorting, check that the ids are in the same order
    for (int i = 0; i < nb_images; ++i)
    {
        std::cout << "Image #" << images[i].to_sort.id << " total : " << images[i].to_sort.total << std::endl;
        std::ostringstream oss;
        oss << "Image#" << images[i].to_sort.id << ".pgm";
        std::string str = oss.str();
        images[i].write(str);
    }

    pipeline.set_images(images);

    std::cout << "Done, the internet is safe now :)" << std::endl;

    // Cleaning
    // TODO : Don't forget to update this if you change allocation style
    for (int i = 0; i < nb_images; ++i)
        images[i].buffer.clear();

    return 0;   
}
#include "hip/hip_runtime.h"
#include "histogram.cuh"

///////////////////////// histogram 1 ///////////////////////////

__global__ void compute_histogram1(int* buffer, int* histogram, int size)
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < size)
    {
        atomicAdd(&histogram[buffer[gid]], 1);
    }
}

///////////////////////// histogram 2 ///////////////////////////

__global__ void compute_histogram2(int* buffer, int* histogram, int size)
{
    constexpr int bin_size = 1 << 8;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    
    __shared__ int shared_histogram[bin_size];

    for (int i = threadIdx.x; i < bin_size; i += blockDim.x)
        shared_histogram[i] = 0;

    __syncthreads();

    if (gid < size)
        atomicAdd(&shared_histogram[buffer[gid]], 1);

    __syncthreads();

    for (int i = threadIdx.x; i < bin_size; i += blockDim.x)
        atomicAdd(&histogram[i], shared_histogram[i]);
}
#include "hip/hip_runtime.h"
#include "radix_sort.cuh"

__global__ void order_checking(int* d_reduce, int* d_total, int size)
{
    extern __shared__ int s_arr[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    if (gid < size)
    {
        s_arr[tid] = d_reduce[gid];
    }
    else
    {
        s_arr[tid] = 0;
    }

    __syncthreads();

    // Perform order checking
    if (gid < (size - 1))
    {
        s_arr[gid] = s_arr[gid] > s_arr[gid + 1];
    }
    s_arr[size - 1] = 0;
    __syncthreads();

    // Perform optimized reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_arr[tid] += s_arr[tid + offset];
        }
        __syncthreads();
    }

    // Write the block sum to global memory
    if (tid == 0)
    {
        atomicAdd(d_total, s_arr[0]);
    }
}

__global__ void radix_sort(int* d_arr_in, int* d_blocks_sum, int* d_prefix_sum, int bit_shift, int* d_arr_out, int size)
{
    extern __shared__ int s_arr[];

    int* s_data = s_arr;
    int* s_mask = s_arr + blockDim.x;
    int* s_local_prefix_sum = s_arr + blockDim.x + blockDim.x + 1;
    int* s_mask_scan = s_arr + blockDim.x + blockDim.x + 1 + blockDim.x;

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    if (gid < size)
    {
        s_data[tid] = d_arr_in[gid];
    }
    else
    {
        s_data[tid] = 0;
    }

    __syncthreads();

    unsigned int data = s_data[tid];
    unsigned int bits = (data >> bit_shift) & 3;

    for (int b = 0; b <= 3; b++)
    {
        // Create the mask
        s_mask[tid] = 0;
        __syncthreads();
        bool is_equal = false;
        if (gid < size)
        {
            is_equal = (bits == b);
            s_mask[tid] = is_equal;
        }
        __syncthreads();

        // Perform scan on the mask
        for (int s = 1; s <= blockDim.x / 2; s <<= 1)
        {
            if (tid > s)
                s_mask[tid] += s_mask[tid - s];
            __syncthreads();    
        }

        //__syncthreads();

        // Shift to the right to produce an exclusive prefix sum
        s_mask[tid + 1] = s_mask[tid];

        __syncthreads();

        if (tid == 0)
        {
            s_mask[0] = 0;
            int total = s_mask[blockDim.x];
            int block_index = gridDim.x * b + blockIdx.x;
            d_blocks_sum[block_index] = total;

            s_mask_scan[b] = total;
        }

        __syncthreads();

        if (is_equal && gid < size)
        {
            s_local_prefix_sum[tid] = s_mask[tid];
        }
    }

    // Perform scan on the d_mask_scan
    for (int s = 1; s <= blockDim.x / 2; s <<= 1)
    {
        if (tid > s)
            s_mask_scan[tid] += s_mask_scan[tid - s];
        __syncthreads();    
    }

    // Shift to the right to produce an exclusive prefix sum
    s_mask_scan[tid + 1] = s_mask_scan[tid];

    __syncthreads();

    if (tid == 0) s_mask_scan[0] = 0;

    if (gid < size)
    {
        int t = s_local_prefix_sum[tid];
        int new_position = t + s_mask_scan[bits];
        
        __syncthreads();

        s_data[new_position] = data;
        s_local_prefix_sum[new_position] = t;

        __syncthreads();

        d_prefix_sum[gid] = s_local_prefix_sum[tid];
        d_arr_out[gid] = s_data[tid];
    }
}

__global__ void compute_new_position(int *d_arr_out, int* d_arr_in, int* d_prefix_sum, int* d_scan_blocks_sum, int bit_shift, int size)
{
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    if (gid >= size) return;

    int data = d_arr_out[gid];
    int bits = (data >> bit_shift) & 3;
    int m = d_prefix_sum[gid];
    int final_index = bits * gridDim.x + blockIdx.x;
    int final_position = d_scan_blocks_sum[final_index] + m;
    __syncthreads();
    d_arr_in[final_position] = data;
}

void radix_sort_gpu(int* arr, int size)
{
    int block_size = 4;
    int grid_size = (size + block_size - 1) / block_size;

    int shared_memory_size = sizeof(int) * (
        block_size + (block_size + 1) + block_size + 8
    );

    int* d_arr_in;
    hipMalloc((void**)&d_arr_in, size * sizeof(int));
    hipMemset(d_arr_in, 0, size * sizeof(int));
    hipMemcpy(d_arr_in, arr, size * sizeof(int), hipMemcpyHostToDevice);

    int* d_arr_out;
    hipMalloc((void**)&d_arr_out, size * sizeof(int));
    hipMemset(d_arr_out, 0, size * sizeof(int));

    int* d_reduce;
    hipMalloc((void**)&d_reduce, size * sizeof(int));
    hipMemset(d_reduce, 0, size * sizeof(int));
    hipMemcpy(d_reduce, arr, size * sizeof(int), hipMemcpyHostToDevice);

    int* d_total;
    hipMalloc((void**)&d_total, sizeof(int));
    hipMemset(d_total, 0, sizeof(int));

    int* d_blocks_sum;
    hipMalloc((void**)&d_blocks_sum, 4 * grid_size * sizeof(int));
    hipMemset(d_blocks_sum, 0, 4 * grid_size * sizeof(int));

    int* d_prefix_sum;
    hipMalloc((void**)&d_prefix_sum, size * sizeof(int));
    hipMemset(d_prefix_sum, 0, size * sizeof(int));

    int* d_scan_blocks_sum;
    hipMalloc((void**)&d_scan_blocks_sum, size * sizeof(int));
    hipMemset(d_scan_blocks_sum, 0, size * sizeof(int));

    int* d_global_counter;
    hipMalloc((void**)&d_global_counter, sizeof(int));

    int* d_blocks_aggregate;
    hipMalloc((void**)&d_blocks_aggregate, grid_size * sizeof(int));
    
    cuda::std::atomic<char>* d_block_states;
    hipMalloc((void**)&d_block_states, grid_size * sizeof(cuda::std::atomic<int>));

    int shared_mem_size = block_size * sizeof(int);

    for (unsigned int bit = 0; bit <= 30; bit += 2)
    {
        // Perform order checking on the current stage
        order_checking<<<grid_size, block_size, shared_mem_size>>>(d_reduce, d_total, size);
        hipDeviceSynchronize();

        // Check if the array is already sorted
        int total;
        hipMemcpy(&total, d_total, sizeof(int), hipMemcpyDeviceToHost);
        if (total == 0)
        {
            std::cout << "Array is already sorted" << std::endl;
            break;
        }

        // Perform radix sort
        radix_sort<<<grid_size, block_size, shared_memory_size>>>(d_arr_in, d_blocks_sum, d_prefix_sum, bit, d_arr_out, size);
        hipDeviceSynchronize();

        hipMemset(d_global_counter, 0, sizeof(int));
        hipMemset(d_block_states, 'X', grid_size * sizeof(char));
        hipMemset(d_blocks_aggregate, 0, grid_size * sizeof(int));

        // Perform scan on the blocks sum
        decoupled_look_back_optimized<<<grid_size, block_size, sizeof(int)>>>(d_blocks_sum, 4 * grid_size, d_blocks_aggregate, d_global_counter, d_block_states);
        hipDeviceSynchronize();

        // Shift for an exclusive prefix sum
        shift_buffer<<<grid_size, block_size>>>(d_blocks_sum, d_scan_blocks_sum, 4 * grid_size);
        hipDeviceSynchronize();
        
        // Compute the new position of each element
        compute_new_position<<<grid_size, block_size>>>(d_arr_out, d_arr_in, d_prefix_sum, d_scan_blocks_sum, bit, size);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr_in, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr_in);
    hipFree(d_blocks_sum);
    hipFree(d_prefix_sum);
    hipFree(d_scan_blocks_sum);
    hipFree(d_reduce);
    hipFree(d_total);
    hipFree(d_arr_out);
    hipFree(d_global_counter);
    hipFree(d_block_states);
}